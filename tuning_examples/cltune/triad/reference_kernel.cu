
#include <hip/hip_runtime.h>

// Select which precision that are used in the calculations
#if PRECISION == 32
    #define DATA_TYPE float
#elif PRECISION == 64
    #define DATA_TYPE double
#endif

// ****************************************************************************
// Function: triad
//
// Purpose:
//   A simple vector addition kernel
//   C = A + s*B
//
// Arguments:
//   A,B - input vectors
//   C - output vectors
//   s - scalar
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: December 15, 2009
//
// Modifications:
//
// ****************************************************************************
extern "C" __global__ void triad_f(float* A, float* B, float* C, float s, int numberOfElements)
{
    int gid = threadIdx.x + (blockIdx.x * blockDim.x);
    
    // Ensure that the current thread id is less than total number of elements
    if (gid < numberOfElements) {
        C[gid] = A[gid] + s*B[gid];
    }
}

extern "C" __global__ void triad_d(double* A, double* B, double* C, double s, int numberOfElements)
{
    int gid = threadIdx.x + (blockIdx.x * blockDim.x);
    
    // Ensure that the current thread id is less than total number of elements
    if (gid < numberOfElements) {
        C[gid] = A[gid] + s*B[gid];
    }
}

extern "C" __global__ void triad_helper(float* Af, float* Bf, float* Cf, float sf, double* Ad, double* Bd, double* Cd, double sd, int numberOfElements) {
    #if PRECISION == 32
        triad_f(Af, Bf, Cf, sf, numberOfElements);
    #elif PRECISION == 64
        triad_d(Ad, Bd, Cd, sd, numberOfElements);
    #endif
}